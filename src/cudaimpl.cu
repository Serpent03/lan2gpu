#include "hip/hip_runtime.h"
#include "../include/common.h"
#include "../include/gpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__ void _cuda_mat_add(float *mat1, float *mat2, float *out,
                              size_t size) {
  int i = blockDim.x * blockIdx.x +
          threadIdx.x; /* big things come in small measures */
  if (i < size) {
    out[i] = mat1[i] + mat2[i];
  }
}

__global__ void _cuda_mat_transpose(float *mat, float *out, size_t size) {
  /* some magic stuff goes in here. lmao */
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size) {
    out[i] = mat[i] + 0.5;
  }
}

extern "C" void cuda_mat_add(float *mat1, float *mat2, float *res, int r,
                             int c) {
  /* d_a and d_b are both the input buffers. d_c is generated in the GPU memory
  and copied over to res, which is then returned for further processing. */
  float *d_a, *d_b, *d_out;
  hipMalloc((void **)&d_a, r * c * sizeof(float));
  hipMalloc((void **)&d_b, r * c * sizeof(float));
  hipMalloc((void **)&d_out, r * c * sizeof(float));

  hipMemcpy(d_a, mat1, r * c * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, mat2, r * c * sizeof(float), hipMemcpyHostToDevice);

  /* threadCount limit of ~1024. Can't launch with just that - we can use the
  2^31 - 1 blocks instead... :) we first take the total size {r * c}, and
  because dividing it by 256 might not cover all the cases, add one. eg: row,
  col = 100, 100 => 10000 separate elements 10000 / 256(threads per block) =>
  ~39; 39 * 256 = 9984 different threads -- which does **not** cover all the
  separate elements.

  So:
  => 10000 / 256 + 255 / 256
  => ~(39 + 0.9)
  => ~(40.8)
  => ~40; 40 * 256 = 10240, which is enough to cover all elements. */
  // uint16 THREADS_PER_BLOCK = 256;
  int32 blockCount = (r * c + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  _cuda_mat_add<<<blockCount, THREADS_PER_BLOCK>>>(d_a, d_b, d_out, r * c);

  hipMemcpy(res, d_out, r * c * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);
}

extern "C" void cuda_mat_transpose(float *mat, float *res, int r, int c) {
  float *d_a, *d_out;
  hipMalloc((void **)&d_a, r * c * sizeof(float));
  hipMalloc((void **)&d_out, r * c * sizeof(float));

  hipMemcpy(d_a, mat, r * c * sizeof(float), hipMemcpyHostToDevice);

  int32 blockCount = (r * c + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  _cuda_mat_transpose<<<blockCount, THREADS_PER_BLOCK>>>(d_a, d_out, r * c);

  hipMemcpy(res, d_out, r * c * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_out);
}